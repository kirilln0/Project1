#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <stdio.h>

#include <iostream>
#include <iomanip>
#include <cmath>
#include <chrono>

// Input size
int const BATCH = 1; //Must be 1 in this program
int const DEPTH = 1;
int const WIDTH = 128;
int const LENGTH = 128;
// Kernel characteristics
int const ZPADX = 0;
int const ZPADY = 0;
int const STRIDEX = 1;
int const STRIDEY = 1;
int const CONV_RECP_SIZEX = 3;
int const CONV_RECP_SIZEY = 3;
int const NUM_OF_KERNELS = 1;
// Convolution output characteristics
int const convLayerSizeX = ((WIDTH - CONV_RECP_SIZEX + 2 * ZPADX) / STRIDEX + 1);
int const convLayerSizeY = ((LENGTH - CONV_RECP_SIZEY + 2 * ZPADY) / STRIDEY + 1);
// transformation matrix characteristics
int const transformSizeY = convLayerSizeY * convLayerSizeX;
int const transformSizeX = CONV_RECP_SIZEX * CONV_RECP_SIZEY * DEPTH;

__global__
void transformToMul(float* inputMatrix, float* reducedMatrix)
{

	int Y = blockIdx.y * blockDim.y + threadIdx.y;
	int X = blockIdx.x * blockDim.x + threadIdx.x;

	if (Y < transformSizeY)
	{
		int inputX = (Y % convLayerSizeX) * STRIDEX + X % CONV_RECP_SIZEY;
		int inputY = (Y / convLayerSizeX) * STRIDEY + (X % (CONV_RECP_SIZEX * CONV_RECP_SIZEY)) / CONV_RECP_SIZEX;
		int inputZ = X / (CONV_RECP_SIZEX * CONV_RECP_SIZEY);
		if ((inputX >= ZPADX && inputX <= (ZPADX + WIDTH - 1)) && (inputY >= ZPADY && inputY <= (ZPADY + LENGTH - 1)))
		{

			reducedMatrix[(Y * transformSizeX) + X] = inputMatrix[(inputZ * LENGTH + inputY - ZPADY) * WIDTH + inputX - ZPADX];
		}
		else
		{
			reducedMatrix[(Y * transformSizeX) + X] = 0;
		}
	}
}

void generateFlat4DData(float* matrix, int x, int y, int z, int d, double type, double jump)
{
	double w = jump;
	for (int b = 0; b < d; b++)
	{
		for (int c = 0; c < z; c++)
		{
			std::cout << "slice: " << c + 1 << "\n";
			for (int j = 0; j < y; j++)
			{
				for (int i = 0; i < x; i++)
				{
					if (type == -1)
					{
						matrix[((b * z + c) * y + j) * x + i] = rand() % 10;
					}
					else if (type == 0)
					{
						matrix[((b * z + c) * y + j) * x + i] = jump;
					}
					else
					{
						matrix[((b * z + c) * y + j) * x + i] = w;
						w += jump;
					}

					std::cout << std::setprecision(1) << std::fixed << matrix[((b * z + c) * y + j) * x + i] << " , ";
				}
				std::cout << "\n";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}
}

int main()
{
	// Performance test variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipError_t cudaStatus;

	// Initialize Host data, kernel and output
	float* hostInputMatrix = new float[BATCH * DEPTH * LENGTH * WIDTH];
	float* hostTransformedInput = new float[transformSizeY * transformSizeX]();

	// GENERATING INPUT
	std::cout << "Inputs:\n";
	generateFlat4DData(hostInputMatrix, WIDTH, LENGTH, DEPTH, BATCH, 1, 0.1);

	// Initializing and allocating Device data, kernels and output
	float* deviceInputMatrix;
	float* deviceTransformedInput;

	cudaStatus = hipMalloc((void **)&deviceInputMatrix, (DEPTH * LENGTH * WIDTH) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void **)&deviceTransformedInput, (transformSizeY * transformSizeX) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(deviceInputMatrix, hostInputMatrix, (DEPTH * LENGTH * WIDTH) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Initializing sizes of grid and block of threads 
	dim3 threadsPerBlock(transformSizeX, transformSizeY);
	dim3 blocksPerGrid(1, 1);
	if (transformSizeY * transformSizeX > 1024) {
		threadsPerBlock.x = transformSizeX;
		threadsPerBlock.y = 1024 / transformSizeX;
		blocksPerGrid.x = ceil(double(transformSizeX) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(transformSizeY) / double(threadsPerBlock.y));
	}

	// Run the kernel function and meassure time
	hipEventRecord(start, 0);

	transformToMul << < blocksPerGrid, threadsPerBlock >> > (deviceInputMatrix, deviceTransformedInput);
	cudaStatus = cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipEventRecord(stop, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "EventRecord failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "EventSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipEventElapsedTime(&time, start, stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ElapsedTime failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	time = time * 1000;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "DeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Get the results from device
	cudaStatus = hipMemcpy(hostTransformedInput, deviceTransformedInput, (transformSizeX * transformSizeY) * sizeof(float), hipMemcpyDeviceToHost); // Not relevant to this program
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// PRINTING RESULTS
	std::cout << "Transformed matrix:\n";
	for (int k = 0; k < transformSizeY; k++)
	{
		for (int j = 0; j < transformSizeX; j++)
		{

			std::cout << std::setprecision(1) << std::fixed << hostTransformedInput[k * transformSizeX + j] << " ";

		}
		std::cout << "\n";
	}

	// CLEAN UP
	printf("Time for the kernel: %f us\n", time);
Error:
	hipFree(deviceInputMatrix);
	hipFree(deviceTransformedInput);

	return 0;
}
