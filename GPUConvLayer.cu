
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <iostream>
#include <iomanip>
#include <cmath>
#include <chrono>
// Input size
int const BATCH = 1;
int const DEPTH = 3;
int const WIDTH = 1000;
int const LENGTH = 1000;
// Kernel characteristics
int const ZPADX = 0;
int const ZPADY = 0;
int const STRIDEX = 1;
int const STRIDEY = 1;
int const CONV_RECP_SIZEX = 3;
int const CONV_RECP_SIZEY = 3;
int const NUM_OF_KERNELS = 50;
int const convLayerSizeX = ((WIDTH - CONV_RECP_SIZEX + 2 * ZPADX) / STRIDEX + 1);
int const convLayerSizeY = ((LENGTH - CONV_RECP_SIZEY + 2 * ZPADY) / STRIDEY + 1);



__global__
void conv(float* inputm, float* weights, float* outputm ) 
{
	int ROW = (blockIdx.y * blockDim.y + threadIdx.y) % (convLayerSizeX * convLayerSizeY);
	int COL = blockIdx.x * blockDim.x + threadIdx.x;
	int DEP = (blockIdx.y * blockDim.y + threadIdx.y) / (convLayerSizeX * convLayerSizeY);
	for (int i = 0; i < DEPTH; i++)
	{
		for (int j = 0; j < CONV_RECP_SIZEY; j++)
		{
			for (int l = 0; l < CONV_RECP_SIZEX; l++)
			{
				outputm[DEP * convLayerSizeY * convLayerSizeX + ROW * convLayerSizeX + COL] += inputm[i * WIDTH * LENGTH + (j + ROW * STRIDEY) * WIDTH + (l + COL * STRIDEX)] * weights[DEP * DEPTH * CONV_RECP_SIZEX * CONV_RECP_SIZEY + i * CONV_RECP_SIZEX * CONV_RECP_SIZEY + j * CONV_RECP_SIZEX + l];
			}
		}
	}
	
}

int main()
{

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipError_t cudaStatus;

	float* hinputMatrix = new float [BATCH * DEPTH * LENGTH * WIDTH];
	float* hconvLayer = new float[NUM_OF_KERNELS * convLayerSizeY * convLayerSizeX];
	float* hconvLayerWeights = new float[NUM_OF_KERNELS * DEPTH * CONV_RECP_SIZEY * CONV_RECP_SIZEX];
	//GENERATING INPUT
	//std::cout << "Inputs:\n";
	float x = 0;
	for (int b = 0; b < BATCH; b++)
	{
		for (int c = 0; c < DEPTH; c++)
		{
			//std::cout << "slice: " << c + 1 << "\n";
			for (int j = 0; j < LENGTH; j++)
			{
				for (int i = 0; i < WIDTH; i++)
				{
					hinputMatrix[b * DEPTH * LENGTH *WIDTH + c * LENGTH * WIDTH + j * WIDTH + i] = x;
					x += 0.5;
					//std::cout << std::setprecision(1) << std::fixed << hinputMatrix[b * DEPTH * LENGTH *WIDTH + c * LENGTH * WIDTH + j * WIDTH + i]<<" ";//<< " ("<< b * DEPTH * LENGTH *WIDTH + c * LENGTH * WIDTH + j * WIDTH + i<<") ";
				}
				//std::cout << "\n";
			}
			//std::cout << "\n";
		}
		//std::cout << "\n";
	}
	//std::cout << "Weights:\n";
	float w = 0;
	for (int d = 0; d < NUM_OF_KERNELS; d++)
	{
		//std::cout << "kernel: " << d + 1 << "\n";
		for (int c = 0; c < DEPTH; c++)
		{
			//std::cout << "slice: " << c + 1 << "\n";
			for (int j = 0; j < CONV_RECP_SIZEY; j++)
			{
				for (int i = 0; i < CONV_RECP_SIZEX; i++)
				{
					hconvLayerWeights[d * DEPTH * CONV_RECP_SIZEX * CONV_RECP_SIZEY + c * CONV_RECP_SIZEX * CONV_RECP_SIZEY + j * CONV_RECP_SIZEX + i] = w;
					w += 0.1;
					//std::cout << std::setprecision(1) << std::fixed << hconvLayerWeights[d * DEPTH * CONV_RECP_SIZEX * CONV_RECP_SIZEY + c * CONV_RECP_SIZEX * CONV_RECP_SIZEY + j * CONV_RECP_SIZEX + i] << " ";// " (" << d * DEPTH * CONV_RECP_SIZEX * CONV_RECP_SIZEY + c * CONV_RECP_SIZEX * CONV_RECP_SIZEY + j * CONV_RECP_SIZEX + i << ") ";
				}
				//std::cout << "\n";
			}
			//std::cout << "\n";
		}
		//std::cout << "\n";
	}
	//
	float* dinputMatrix;
	float* dconvLayerWeights;
	float* dconvLayer;

	hipMalloc((void **)&dconvLayer, (NUM_OF_KERNELS * convLayerSizeY * convLayerSizeX) * sizeof(float));
	hipMalloc((void **)&dconvLayerWeights, (NUM_OF_KERNELS * DEPTH * CONV_RECP_SIZEY * CONV_RECP_SIZEX) * sizeof(float));
	hipMalloc((void **)&dinputMatrix, (DEPTH * LENGTH * WIDTH) * sizeof(float));


	hipMemcpy(dinputMatrix, hinputMatrix, (DEPTH * LENGTH * WIDTH) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dconvLayerWeights, hconvLayerWeights, (NUM_OF_KERNELS * DEPTH * CONV_RECP_SIZEY * CONV_RECP_SIZEX) * sizeof(float), hipMemcpyHostToDevice);
	

	dim3 threadsPerBlock(convLayerSizeX, convLayerSizeY *  NUM_OF_KERNELS);
	dim3 blocksPerGrid(1, 1);
	if (NUM_OF_KERNELS * convLayerSizeY * convLayerSizeX> 1024) {
		threadsPerBlock.x = 32;
		threadsPerBlock.y = 32;
		blocksPerGrid.x = ceil(double(convLayerSizeX) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(convLayerSizeY *  NUM_OF_KERNELS) / double(threadsPerBlock.y));
	}
	hipEventRecord(start, 0);
	conv<<< blocksPerGrid, threadsPerBlock >>> (dinputMatrix, dconvLayerWeights, dconvLayer);
	cudaStatus = hipEventRecord(stop, 0);
	cudaStatus = hipEventSynchronize(stop);
	hipDeviceSynchronize();
	cudaStatus = hipEventElapsedTime(&time, start, stop);
	time = time * 1000;

	hipMemcpy(hconvLayer, dconvLayer, (NUM_OF_KERNELS * convLayerSizeY * convLayerSizeX) * sizeof(float), hipMemcpyDeviceToHost);
	// PRINTING RESULTS
	std::cout << "Results:\n" << "Convolution matrix:\n";
	for (int k = 0; k < NUM_OF_KERNELS; k++)
	{
		//std::cout << "slice: " << k + 1 << "\n";
		for (int j = 0; j < convLayerSizeY; j++)
		{
			for (int i = 0; i < convLayerSizeX; i++)
			{
				//std::cout << std::setprecision(2) << std::fixed << hconvLayer[k * convLayerSizeY * convLayerSizeX + j * convLayerSizeX + i] << " ";
			}
			//std::cout << "\n";
		}
		//std::cout << "\n";
	}
	// CLEAN UP
	printf("Time for the kernel: %f us\n", time);

	hipFree(dconvLayerWeights);
	hipFree(dinputMatrix);
	hipFree(dconvLayer);
return 0;
}
