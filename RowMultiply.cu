#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include <stdio.h>

#include <iostream>
#include <iomanip>
#include <cmath>
#include <chrono>

// Input size
int const WIDTH = 3*3*3;
int const LENGTHA = 3;
int const LENGTHB = 126*126*3;
// transformation matrix characteristics
int const OUTPUT_SIZEY = LENGTHA * LENGTHB;


__global__
void rowMul(float* A, float* B, float* C)
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int N = X % LENGTHB;
	int n = X / LENGTHB;
	float sum = 0;
	if (X < OUTPUT_SIZEY)
	{
		for (int i = 0; i < WIDTH; i++)
		{
			sum += A[n * WIDTH + i] * B[N * WIDTH + i];
		}
		C[X] = sum;
	}
}



void generateFlat4DData(float* matrix, int x, int y, int z, int d, double type, double jump)
{
	double w = jump;
	for (int b = 0; b < d; b++)
	{
		for (int c = 0; c < z; c++)
		{
			//std::cout << "slice: " << c + 1 << "\n";
			for (int j = 0; j < y; j++)
			{
				for (int i = 0; i < x; i++)
				{
					if (type == -1)
					{
						matrix[((b * z + c) * y + j) * x + i] = rand() % 10;
					}
					else if (type == 0)
					{
						matrix[((b * z + c) * y + j) * x + i] = jump;
					}
					else
					{
						matrix[((b * z + c) * y + j) * x + i] = w;
						w += jump;
					}

					//std::cout << std::setprecision(1) << std::fixed << matrix[((b * z + c) * y + j) * x + i] << " , ";
				}
				//std::cout << "\n";
			}
			//std::cout << "\n";
		}
		//std::cout << "\n";
	}
}

int main()
{
	// Performance test variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipError_t cudaStatus;

	// Initialize Host data, kernel and output
	float* hostInputMatrixA = new float[LENGTHA * WIDTH];
	float* hostInputMatrixB = new float[LENGTHB * WIDTH];
	float* hostResult = new float[OUTPUT_SIZEY]();

	// GENERATING INPUT
	std::cout << "Inputs:\n";
	generateFlat4DData(hostInputMatrixA, WIDTH, LENGTHA, 1, 1, 1, 0.1);
	generateFlat4DData(hostInputMatrixB, WIDTH, LENGTHB, 1, 1, 1, 0.1);

	// Initializing and allocating Device data, kernels and output
	float* deviceInputMatrixA;
	float* deviceInputMatrixB;
	float* deviceResult;

	cudaStatus = hipMalloc((void **)&deviceInputMatrixA, (LENGTHA * WIDTH) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void **)&deviceInputMatrixB, (LENGTHB * WIDTH) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void **)&deviceResult, (OUTPUT_SIZEY) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(deviceInputMatrixA, hostInputMatrixA, (LENGTHA * WIDTH) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(deviceInputMatrixB, hostInputMatrixB, (LENGTHA * WIDTH) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Initializing sizes of grid and block of threads 
	dim3 threadsPerBlock(OUTPUT_SIZEY);
	dim3 blocksPerGrid(1);
	if (OUTPUT_SIZEY > 1024) {
		threadsPerBlock.x = 1024;
		blocksPerGrid.x = ceil(double(OUTPUT_SIZEY) / double(threadsPerBlock.x));
	}

	// Run the kernel function and meassure time
	hipEventRecord(start, 0);

	rowMul << < blocksPerGrid, threadsPerBlock >> > (deviceInputMatrixA, deviceInputMatrixB, deviceResult);
	cudaStatus = hipEventRecord(stop, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "EventRecord failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "EventSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipEventElapsedTime(&time, start, stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ElapsedTime failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	time = time * 1000;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "DeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Get the results from device
	cudaStatus = hipMemcpy(hostResult, deviceResult, (OUTPUT_SIZEY) * sizeof(float), hipMemcpyDeviceToHost); // Not relevant to this program
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// PRINTING RESULTS
	std::cout << "Transformed matrix:\n";
	for (int k = 0; k < OUTPUT_SIZEY; k++)
	{
			//std::cout << std::setprecision(1) << std::fixed << hostResult[k] << " , ";
	}
	printf("\n");
	// CLEAN UP
	printf("Time for the kernel: %f us\n", time);
Error:
	hipFree(deviceInputMatrixA);
	hipFree(deviceInputMatrixB);
	hipFree(deviceResult);

	return 0;
}
